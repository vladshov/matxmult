// Test File read.cpp : Defines the entry point for the console application.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

#include <math.h>
#include <ctime>

using namespace std;

int main (int argc, char *argv[])
{
	ifstream in_stream;
	in_stream.open(argv[1]);
//	in_stream.open("D:/1.txt");
	int m1;
	int n1;
	in_stream >> m1;
	in_stream >> n1;
	double **A = new double* [m1];
	int i, j;
	for (i = 0; i < m1; i++) {
		(A)[i] = new double [n1];
		for (j = 0; j < n1; j++) {
			in_stream >> A[i][j];
		}
	}
	int m2;
	int n2;
	in_stream >> m2;
	in_stream >> n2;
	double **B = new double* [m2];

	for (i = 0; i < m2; i++) {
		(B)[i] = new double [n2];
		for (j = 0; j < n2; j++) {
			in_stream >> B[i][j];
		}
	}
	in_stream.close();
	int k;
	double **C = new double* [m1];
	for (i = 0; i < m1; i++) {
		(C)[i] = new double [n2];
		for (j = 0; j < n2; j++) {
			C[i][j] = 0;
			for (k = 0; k < n1; k++) {
				C[i][j] += A[i][k]*B[k][j];
			}
		}
	}

	ofstream fs(argv[2]);
//	ofstream fs("D:/2.txt");

    if(!fs)
    {
        cerr<<"Cannot open the output file."<<endl;
        return 1;
    }
	
	for (i = 0; i < m1; i++) {
		for (j = 0; j < n2; j++) {
			fs<< C[i][j];
			fs<< " ";
		}
		fs<< "\n";
	}

    fs.close();
	return 0;
}



